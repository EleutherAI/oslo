#include "hip/hip_runtime.h"
/* Copyright 2019 The Microsoft DeepSpeed Team */
#include "ATen/ATen.h"
#include "ATen/TensorUtils.h"
#include "ATen/cuda/HIPContext.h"
#include "ATen/cuda/detail/IndexUtils.cuh"
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include "ATen/Type.h"
#include "ATen/AccumulateType.h"

#include <iostream>

//#include <helper_functions.h>
#if defined(__HIP_PLATFORM_HCC__) && HIP_VERSION > 305
#include <hip/hip_cooperative_groups.h>
#else
#include <hip/hip_cooperative_groups.h>
#endif
#include <hip/hip_runtime_api.h>
#include <stdio.h>

namespace cg = cooperative_groups;

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
namespace {
// This is the un-specialized struct.  Note that we prevent instantiation of
// this struct by putting an undefined symbol in the function body so it won't
// compile.
template <typename T> struct SharedMemory {
  // Ensure that we won't compile any un-specialized types
  __device__ inline operator T *() {
#ifndef _WIN32
    extern __device__ void error(void);
    error();
#endif
    return NULL;
  }
};

template <> struct SharedMemory<float> {
  __device__ inline operator float *() {
    extern __shared__ float s_float[];
    return s_float;
  }
};

template <> struct SharedMemory<double> {
  __device__ inline operator double *() {
    extern __shared__ double s_double[];
    return s_double;
  }
};
} // namespace

#include "type_shim.h"

typedef enum {
  ADAM_MODE_0 = 0, // eps under square root
  ADAM_MODE_1 = 1  // eps outside square root
} adamMode_t;

// s_a and s_b are in shared memory
// g_a and g_b are in shared memory
template <typename T, int blockSize>
__device__ void reduce_block_in_shared_memory(T *s_a, T *s_b, T *g_a, T *g_b) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();

  // perform block reduction in shared memory,
  unsigned int tid = cta.thread_rank();

  T a_sum = s_a[tid];
  T b_sum = s_b[tid];

  cg::sync(cta);

  // do reduction in shared mem
  if ((blockSize >= 512) && (tid < 256)) {
    s_a[tid] = a_sum = a_sum + s_a[tid + 256];
    s_b[tid] = b_sum = b_sum + s_b[tid + 256];
  }

  cg::sync(cta);

  if ((blockSize >= 256) && (tid < 128)) {
    s_a[tid] = a_sum = a_sum + s_a[tid + 128];
    s_b[tid] = b_sum = b_sum + s_b[tid + 128];
  }

  cg::sync(cta);

  if ((blockSize >= 128) && (tid < 64)) {
    s_a[tid] = a_sum = a_sum + s_a[tid + 64];
    s_b[tid] = b_sum = b_sum + s_b[tid + 64];
  }

  cg::sync(cta);

#if (__CUDA_ARCH__ >= 300)
  if (tid < 32) {
    cg::coalesced_group active = cg::coalesced_threads();

    // Fetch final intermediate sum from 2nd warp
    if (blockSize >= 64) {
      a_sum = a_sum + s_a[tid + 32];
      b_sum = b_sum + s_b[tid + 32];
    }

    // Reduce final warp using shuffle
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
      a_sum += active.shfl_down(a_sum, offset);
      b_sum += active.shfl_down(b_sum, offset);
    }
  }
#else
  if ((blockSize >= 64) && (tid < 32)) {
    s_a[tid] = a_sum = a_sum + s_a[tid + 32];
    s_b[tid] = b_sum = b_sum + s_b[tid + 32];
  }

  cg::sync(cta);

  if ((blockSize >= 32) && (tid < 16)) {
    s_a[tid] = a_sum = a_sum + s_a[tid + 16];
    s_b[tid] = b_sum = b_sum + s_b[tid + 16];
  }

  cg::sync(cta);

  if ((blockSize >= 16) && (tid < 8)) {
    s_a[tid] = a_sum = a_sum + s_a[tid + 8];
    s_b[tid] = b_sum = b_sum + s_b[tid + 8];
  }

  cg::sync(cta);

  if ((blockSize >= 8) && (tid < 4)) {
    s_a[tid] = a_sum = a_sum + s_a[tid + 4];
    s_b[tid] = b_sum = b_sum + s_b[tid + 4];
  }

  cg::sync(cta);

  if ((blockSize >= 4) && (tid < 2)) {
    s_a[tid] = a_sum = a_sum + s_a[tid + 2];
    s_b[tid] = b_sum = b_sum + s_b[tid + 2];
  }

  cg::sync(cta);

  if ((blockSize >= 2) && (tid < 1)) {
    s_a[tid] = a_sum = a_sum + s_a[tid + 1];
    s_b[tid] = b_sum = b_sum + s_b[tid + 1];
  }

  cg::sync(cta);

#endif

  // write result for this block to global mem
  if (tid == 0) {
    g_a[blockIdx.x] = (T)a_sum;
    g_b[blockIdx.x] = (T)b_sum;
  }
}

template <typename T, int blockSize>
__device__ void reduce_two_vectors_in_register(T a, T b, T *g_a, T *g_b) {
  const int threadIdInBlock = cg::this_thread_block().thread_rank();

  T *s_a = SharedMemory<T>();
  T *s_b = SharedMemory<T>() + cg::this_thread_block().size();

  s_a[threadIdInBlock] = a;
  s_b[threadIdInBlock] = b;

  reduce_block_in_shared_memory<T, blockSize>(s_a, s_b, g_a, g_b);
}

template <typename T, typename GRAD_T, int blockSize>
__global__ void lamb_cuda_kernel_part1(
    T *__restrict__ p,
    GRAD_T *__restrict__ p_copy, // For mixed precision training, pass NULL if
                                 // not needed
    T *__restrict__ m, T *__restrict__ v, const GRAD_T *__restrict__ g,
    const float b1, const float b2, const float eps, const float grad_scale,
    const float step_size, const size_t tsize, adamMode_t mode,
    const float decay, T *__restrict__ w_l2_i, T *__restrict__ u_l2_i) {
  // Assuming 2D grids and 2D blocks
  const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
  const int threadsPerBlock = blockDim.x * blockDim.y;
  const int threadIdInBlock = cg::this_thread_block().thread_rank();
  const int i = (blockId * threadsPerBlock + threadIdInBlock);
  const int totThreads = gridDim.x * gridDim.y * threadsPerBlock;

  T reg_w = 0;
  T reg_u = 0;

  for (int j = i; j < tsize; j += totThreads) {
    T scaled_grad = g[j] / grad_scale;
    T pj = p[j];
    m[j] = b1 * m[j] + (1 - b1) * scaled_grad;
    v[j] = b2 * v[j] + (1 - b2) * scaled_grad * scaled_grad;
    float denom;
    if (mode == ADAM_MODE_0)
      denom = sqrtf(v[j] + eps);
    else // Mode 1
      denom = sqrtf(v[j]) + eps;
    T update = (m[j] / denom) + (decay * p[j]);

    reg_u += update * update;
    reg_w += pj * pj;
  }

  reduce_two_vectors_in_register<T, blockSize>(reg_w, reg_u, w_l2_i, u_l2_i);
}

template <typename T, typename GRAD_T, int blockSize>
__global__ void lamb_cuda_kernel_part2(const size_t tsize, T *__restrict__ g_a,
                                       T *__restrict__ g_b) {
  T *s_a = SharedMemory<T>();
  T *s_b = SharedMemory<T>() + cg::this_thread_block().size();

  const int threadIdInBlock = cg::this_thread_block().thread_rank();

  s_a[threadIdInBlock] = g_a[threadIdInBlock];
  s_b[threadIdInBlock] = g_b[threadIdInBlock];

  if (threadIdInBlock >= tsize) {
    s_a[threadIdInBlock] = 0.0;
    s_b[threadIdInBlock] = 0.0;
  }

  reduce_block_in_shared_memory<T, blockSize>(s_a, s_b, g_a, g_b);
}

template <typename T, typename GRAD_T>
__global__ void lamb_cuda_kernel_part3(
    T *__restrict__ p,
    GRAD_T *__restrict__ p_copy, // For mixed precision training, pass NULL if
                                 // not needed
    T *__restrict__ m, T *__restrict__ v, const GRAD_T *__restrict__ g,
    const float b1, const float b2, const float max_coeff,
    const float min_coeff, const float eps, const float grad_scale,
    const float step_size, const size_t tsize, adamMode_t mode,
    const float decay, T *__restrict__ w_l2_i, T *__restrict__ u_l2_i,
    T *__restrict__ lamb_coeff_val) {
  // Assuming 2D grids and 2D blocks
  const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
  const int threadsPerBlock = blockDim.x * blockDim.y;
  const int threadIdInBlock = cg::this_thread_block().thread_rank();
  const int i = (blockId * threadsPerBlock + threadIdInBlock);
  const int totThreads = gridDim.x * gridDim.y * threadsPerBlock;

  T reg_w = sqrtf(w_l2_i[0]);
  T reg_u = sqrtf(u_l2_i[0]);

  float lamb_coeff = 1.0;

  if (reg_w != 0 && reg_u != 0) {
    lamb_coeff = reg_w / reg_u;
    if (lamb_coeff > max_coeff) {
      lamb_coeff = max_coeff;
    }
    if (lamb_coeff < min_coeff) {
      lamb_coeff = min_coeff;
    }
  }

  if (blockId == 0 && threadIdInBlock == 0) {
    lamb_coeff_val[0] = lamb_coeff;
    // printf("Cuda Lamb Coeff is %.6f \n",lamb_coeff);
  }

  for (int j = i; j < tsize; j += totThreads) {
    T pj = (float)p[j];
    T mj = m[j];
    T vj = v[j];
    float denom;
    if (mode == ADAM_MODE_0)
      denom = sqrtf(vj + eps);
    else // Mode 1
      denom = sqrtf(vj) + eps;
    T update = (mj / denom) + (decay * pj);

    pj = pj - (step_size * lamb_coeff * update);
    p[j] = pj;
    if (p_copy != NULL)
      p_copy[j] = (GRAD_T)pj;
  }
}

void fused_lamb_cuda(at::Tensor &p, at::Tensor &p_copy, at::Tensor &m,
                     at::Tensor &v, at::Tensor &g, float lr, float beta1,
                     float beta2, float max_coeff, float min_coeff, float eps,
                     float grad_scale, int step, int mode, int bias_correction,
                     float decay, at::Tensor &w_l2_i, at::Tensor &u_l2_i,
                     at::Tensor &lamb_coeff) {
  //        using namespace at;

  // Get tensor size
  int tsize = p.numel();
  // Determine #threads and #blocks
  const int threadsPerBlock = 512;
  int num_blocks = (tsize + threadsPerBlock - 1) / threadsPerBlock;
  if (num_blocks > 512)
    num_blocks = 512;

  int smemsize = 0;

  if (p.type().scalarType() == at::ScalarType::Double)
    smemsize = 2 * threadsPerBlock * sizeof(double);
  else
    smemsize = 2 * threadsPerBlock * sizeof(float);

  const dim3 blocks(num_blocks);
  const dim3 threads(threadsPerBlock);

  AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p),
             "parameter tensor is too large to be indexed with int32");
  // Constants
  float step_size = 0;
  if (bias_correction == 1) {
    const float bias_correction1 = 1 - std::pow(beta1, step);
    const float bias_correction2 = 1 - std::pow(beta2, step);
    step_size = lr * std::sqrt(bias_correction2) / bias_correction1;
  } else {
    step_size = lr;
  }
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (g.type().scalarType() == at::ScalarType::Half) {
    // all other values should be fp32 for half gradients
    AT_ASSERTM(p.type().scalarType() == at::ScalarType::Float,
               "expected parameter to be of float type");
    // dispatch is done on the gradient type
    using namespace at; // prevents "toString is undefined" errors
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        g.scalar_type(), "lamb_cuda_kernel", ([&] {
          using accscalar_t = at::acc_type<scalar_t, true>;

          lamb_cuda_kernel_part1<accscalar_t, scalar_t, threadsPerBlock>
              <<<blocks, threadsPerBlock, smemsize, stream>>>(
                  p.data<accscalar_t>(),
                  p_copy.numel() ? p_copy.data<scalar_t>() : NULL,
                  m.data<accscalar_t>(), v.data<accscalar_t>(),
                  g.data<scalar_t>(), beta1, beta2, eps, grad_scale, step_size,
                  tsize, (adamMode_t)mode, decay, w_l2_i.data<accscalar_t>(),
                  u_l2_i.data<accscalar_t>());

          lamb_cuda_kernel_part2<accscalar_t, scalar_t, threadsPerBlock>
              <<<1, threadsPerBlock, smemsize, stream>>>(
                  num_blocks, w_l2_i.data<accscalar_t>(),
                  u_l2_i.data<accscalar_t>());

          lamb_cuda_kernel_part3<accscalar_t, scalar_t>
              <<<blocks, threadsPerBlock, smemsize, stream>>>(
                  p.data<accscalar_t>(),
                  p_copy.numel() ? p_copy.data<scalar_t>() : NULL,
                  m.data<accscalar_t>(), v.data<accscalar_t>(),
                  g.data<scalar_t>(), beta1, beta2, max_coeff, min_coeff, eps,
                  grad_scale, step_size, tsize, (adamMode_t)mode, decay,
                  w_l2_i.data<accscalar_t>(), u_l2_i.data<accscalar_t>(),
                  lamb_coeff.data<accscalar_t>());
        }));
  } else {
    using namespace at;
    AT_DISPATCH_FLOATING_TYPES(
        g.scalar_type(), "lamb_cuda_kernel", ([&] {
          lamb_cuda_kernel_part1<scalar_t, scalar_t, threadsPerBlock>
              <<<blocks, threadsPerBlock, smemsize, stream>>>(
                  p.data<scalar_t>(),
                  NULL, // don't output p_copy for fp32, it's wasted write
                  m.data<scalar_t>(), v.data<scalar_t>(), g.data<scalar_t>(),
                  beta1, beta2, eps, grad_scale, step_size, tsize,
                  (adamMode_t)mode, decay, w_l2_i.data<scalar_t>(),
                  u_l2_i.data<scalar_t>());

          lamb_cuda_kernel_part2<scalar_t, scalar_t, threadsPerBlock>
              <<<1, threadsPerBlock, smemsize, stream>>>(
                  num_blocks, w_l2_i.data<scalar_t>(), u_l2_i.data<scalar_t>());

          lamb_cuda_kernel_part3<scalar_t, scalar_t>
              <<<blocks, threadsPerBlock, smemsize, stream>>>(
                  p.data<scalar_t>(),
                  NULL, // don't output p_copy for fp32, it's wasted write
                  m.data<scalar_t>(), v.data<scalar_t>(), g.data<scalar_t>(),
                  beta1, beta2, max_coeff, min_coeff, eps, grad_scale,
                  step_size, tsize, (adamMode_t)mode, decay,
                  w_l2_i.data<scalar_t>(), u_l2_i.data<scalar_t>(),
                  lamb_coeff.data<scalar_t>());
        }));
  }
  C10_CUDA_CHECK(hipGetLastError());
}

// template __device__ void reduce_two_vectors_in_register<float,512>(float a,
// float b, float* g_a, float* g_b, cg::grid_group &cgg);
