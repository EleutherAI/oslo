#include "hip/hip_runtime.h"
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>

#include "../model/decoder.h"
#include "../model/encoder.h"
#include "../proto/transformer_weight.h"
#include "../tools/util.h"

namespace py = pybind11;

#ifdef FP16_MODE
const lightseq::cuda::OperationType decoder_optype =
    lightseq::cuda::OperationType::FP16;
#else
const lightseq::cuda::OperationType decoder_optype =
    lightseq::cuda::OperationType::FP32;
#endif

namespace lightseq {
namespace cuda {
class TransformerDecoder {
 private:
  typedef lightseq::cuda::OperationTypeTraits<decoder_optype> optraits;
  lightseq::cuda::Decoder<decoder_optype> *decoder_;

  optraits::DataType *d_encoder_output_;
  int *d_output_;
  int *d_padding_mask_;
  int _max_batch_size;
  hipStream_t stream_;
  hipblasHandle_t hd_;
  lightseq::cuda::TransformerWeight<decoder_optype> tw_;

 public:
  TransformerDecoder(const std::string weight_path, const int max_batch_size)
      : stream_(nullptr), hd_(nullptr), decoder_(nullptr) {
    /* ---step1. init environment--- */
    _max_batch_size = max_batch_size;
    hipError_t cuerr = hipSetDevice(0);
    if (cuerr != hipSuccess) {
      throw std::runtime_error(hipGetErrorString(cuerr));
    }
    cuerr = hipStreamCreate(&stream_);
    if (cuerr != hipSuccess) {
      throw std::runtime_error(hipGetErrorString(cuerr));
    }
    hipblasStatus_t cublaserr = hipblasCreate(&hd_);
    if (cublaserr != HIPBLAS_STATUS_SUCCESS) {
      throw std::runtime_error("Failed to creat cublas handle ");
    }
    cublaserr = hipblasSetStream(hd_, stream_);
    if (cublaserr != HIPBLAS_STATUS_SUCCESS) {
      throw std::runtime_error("Failed to set stream for cublas handle");
    }

    /* ---step2. load model weights into GPU memory--- */

    // saved in custom proto file
    std::string model_weights_path = weight_path;
    std::string res = tw_.initializing(model_weights_path, true);
    if (!res.empty()) {
      throw std::runtime_error(res);
    }

    if (tw_._sampling_method == "topk" || tw_._sampling_method == "topp") {
      tw_._beam_size = 1;
    }
    tw_.print_model_config();
    /*
      step3. instantiate encoder and decoder, init the gpu memory buffer.
        using thrust vector to avoid manage gpu memory by hand
    */

    // instantiate encoder
    // FIXME: padding mask should be passed from user
    // thrust::device_vector<int> d_padding_mask_ =
    //     std::vector<int>(_max_batch_size * tw_._max_step, 0);
    lightseq::cuda::CHECK_GPU_ERROR(hipMalloc(
        &d_padding_mask_, _max_batch_size * tw_._max_step * sizeof(int)));

    lightseq::cuda::CHECK_GPU_ERROR(hipMalloc(
        &d_encoder_output_, _max_batch_size * tw_._max_step * tw_._hidden_size *
                                sizeof(optraits::DataType)));

    lightseq::cuda::CHECK_GPU_ERROR(hipMalloc(
        &d_output_,
        _max_batch_size * tw_._beam_size * tw_._max_step * sizeof(int)));

    decoder_ = new lightseq::cuda::Decoder<decoder_optype>(
        _max_batch_size, d_padding_mask_, d_encoder_output_, d_output_, tw_,
        stream_, hd_, true);
    res = decoder_->check();
    if (!res.empty()) {
      throw std::runtime_error(res);
    }

    long buf_bytesize = decoder_->compute_buffer_bytesize();
    std::cout << "Allocated " << buf_bytesize / (1024 * 1024)
              << "MB GPU buffer for transformer decoder" << std::endl;

    void *d_buf_;
    // encoder and decoder use the same buffer to save gpu memory useage
    lightseq::cuda::CHECK_GPU_ERROR(
        hipMalloc((void **)&d_buf_, (size_t)buf_bytesize));
    decoder_->init_buffer(d_buf_);
    cuerr = hipStreamSynchronize(stream_);
    if (cuerr != hipSuccess) {
      std::cout << "failed to init GPU for transformer: " << std::endl;
      std::runtime_error(std::string(hipGetErrorString(cuerr)));
    }
  }

  py::array_t<int> infer(
      py::array_t<float, py::array::c_style | py::array::forcecast>
          encoder_output,
      py::array_t<int, py::array::c_style | py::array::forcecast>
          encoder_mask) {
    auto encoder_out = encoder_output.mutable_unchecked<3>();
    auto encoder_mask_out = encoder_mask.mutable_unchecked<2>();
    const float *encoder_output_data = encoder_out.data(0, 0, 0);
    const int *encoder_mask_data = encoder_mask_out.data(0, 0);
    std::vector<optraits::DataType> h_encoder_out(encoder_out.size());
    for (auto i = 0; i < encoder_out.size(); i++) {
      optraits::DataType data;
      if (decoder_optype == lightseq::cuda::OperationType::FP16) {
        data = __float2half_rn(encoder_output_data[i]);
      } else {
        data = encoder_output_data[i];
      }
      h_encoder_out[i] = data;
    }

    lightseq::cuda::CHECK_GPU_ERROR(
        hipMemcpyAsync(d_encoder_output_, h_encoder_out.data(),
                        sizeof(optraits::DataType) * encoder_out.size(),
                        hipMemcpyHostToDevice, stream_));
    lightseq::cuda::CHECK_GPU_ERROR(
        hipMemcpyAsync(d_padding_mask_, encoder_mask_data,
                        sizeof(int) * encoder_mask_out.size(),
                        hipMemcpyHostToDevice, stream_));

    int batch_size = encoder_out.shape(0);
    int batch_seq_len = encoder_out.shape(1);
    decoder_->run_one_infer(batch_size, batch_seq_len);
    int tokens_size = decoder_->_cur_step + 1;
    int beam_size = tw_._beam_size;
    auto tokens = py::array_t<int>({batch_size, beam_size, tokens_size});
    int *tokens_data = tokens.mutable_data(0, 0);
    lightseq::cuda::CHECK_GPU_ERROR(hipMemcpy(tokens_data, d_output_,
                                               sizeof(int) * tokens.size(),
                                               hipMemcpyDeviceToHost));
    return tokens;
  }
};
}  // namespace cuda
}  // namespace lightseq
